#include<stdio.h>
#include<iostream>
#include<math.h>
#include<vector>

// includes CUDA
#include<hip/hip_runtime.h>



// Functions
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
      fprintf(stderr, "GPUassert: %s %s %dn", hipGetErrorString(code), file, line);
      if (abort) { exit(code); }
    }
}

__global__
void calcF(int nAtoms, long nDiff, 
  double *diffPos3D0, double *diffPos3D1, double *diffPos3D2,
  double *atomOccu, int *atomType,
  double *atomPos3D0, double *atomPos3D1, double *atomPos3D2,
  double *atomDW3D0, double *atomDW3D1, double *atomDW3D2,
  double *atomicF,
  double *outFRe, double *outFIm)
{
  long totalThreadsX = (blockDim.x * gridDim.x);
  long totalThreadsY = (blockDim.y * gridDim.y);
  int calcManyX = ceil((double)nDiff / totalThreadsX);
  int calcManyY = ceil((double)nAtoms / totalThreadsY);

  double pipi16 = (16. * M_PI * M_PI);

  for (int threadRunY = 0; threadRunY < calcManyY; ++threadRunY)
  {
    long atomIndex = threadIdx.y + (blockDim.y * blockIdx.y) + threadRunY * totalThreadsY;

    for (int threadRunX = 0; threadRunX < calcManyX; ++threadRunX)
    {
      long diffIndex = threadIdx.x + (blockDim.x * blockIdx.x) + threadRunX * totalThreadsX;
      
      if (atomIndex < nAtoms && diffIndex < nDiff)
      {
        // atomic form factor
        double diffDot = 0.0;
        diffDot += diffPos3D0[diffIndex] * diffPos3D0[diffIndex];
        diffDot += diffPos3D1[diffIndex] * diffPos3D1[diffIndex];
        diffDot += diffPos3D2[diffIndex] * diffPos3D2[diffIndex];
        double atomicFExpFact = 0.0;
        atomicFExpFact = diffDot / pipi16; // Lambda needs to be included here if diffPos3D is not in Q space!
        double atomicFRe = 1.0;
        for (uint index = 0; index < 4; ++index)
        {
          atomicFRe += (atomicF[atomType[atomIndex] * 9 + index] *
               exp(-atomicF[atomType[atomIndex] * 9 + 4 + index] * atomicFExpFact));
        }
        atomicFRe += atomicF[atomType[atomIndex] * 9 + 9];

        // debye-waller factors 
        double debyWaller = 1.0;
        if (atomDW3D0[atomIndex] < 0) { // if DW0 < 0 => inplane and out-of-plane
          if (atomDW3D1[atomIndex] < 0) { // if DW1 < 0 and => iso
            if (atomDW3D2[atomIndex] < 0) { // if DW2 < 0 no DW 
              debyWaller = 1.0;
            }
            else { //  iso debye-waller
              debyWaller *= exp(-atomDW3D0[atomIndex] * diffDot / pipi16);
            }
          }
          else { //  inplane and out-of-plane debye-waller
            debyWaller *= exp(-atomDW3D1[atomIndex] *
                              (diffPos3D0[diffIndex] * diffPos3D0[diffIndex] +
                               diffPos3D1[diffIndex] * diffPos3D1[diffIndex]) /
                              pipi16);
            debyWaller *= exp(-atomDW3D2[atomIndex] * diffPos3D2[diffIndex] *
                              diffPos3D2[diffIndex] / pipi16);
          }
        } 
        else { //  all direction debye-waller
          debyWaller *= exp(-atomDW3D0[atomIndex] * diffPos3D0[diffIndex] *
                            diffPos3D0[diffIndex] / pipi16);
          debyWaller *= exp(-atomDW3D1[atomIndex] * diffPos3D1[diffIndex] *
                            diffPos3D1[diffIndex] / pipi16);
          debyWaller *= exp(-atomDW3D2[atomIndex] * diffPos3D2[diffIndex] *
                            diffPos3D2[diffIndex] / pipi16);
        }

        // might be possible to speed up by summing up block wise and than sum blocks
        double dotProduct = 0.0;
        dotProduct += atomPos3D0[atomIndex] * diffPos3D0[diffIndex];
        dotProduct += atomPos3D1[atomIndex] * diffPos3D1[diffIndex];
        dotProduct += atomPos3D2[atomIndex] * diffPos3D2[diffIndex];
        // double* thisBaseFIm = (double*)((char*)baseFIm + diffIndex * baseFImpitch);
        // atomicAdd(&outFRe[diffIndex], (thisBaseFRe[atomType[atomIndex]] * atomOccu[atomIndex] * cos(dotProduct)) );
        double cosDot = cos(dotProduct);
        double sinDot = sin(dotProduct);
        double resultFRe = atomOccu[atomIndex] * (atomicFRe * cosDot);
        double resultFIm = atomOccu[atomIndex] * (atomicFRe * sinDot);
        atomicAdd(&outFRe[diffIndex], resultFRe );
        atomicAdd(&outFIm[diffIndex], resultFIm );
      }
    }
  }
}


class DiffRun
{
  public:
    long nAtoms; // along thread.y
    long nDiff; // along thread.x
    int nAtomTypes; // number of atomicF

    double  *atomicF;
    int     *atomType;
    double  *atomOccu;
    double  *atomPos3D0;
    double  *atomPos3D1;
    double  *atomPos3D2;
    double  *atomDW3D0;
    double  *atomDW3D1;
    double  *atomDW3D2;
    double  *diffPos3D0;
    double  *diffPos3D1;
    double  *diffPos3D2;
    double  *diffFstaticRe;
    double  *diffFstaticIm;
    double  *diffFunitRe;
    double  *diffFunitIm;

    bool finishedRun = false;
    void run(void);

    // determine compute structure
    int nTx = 64;
    int nTy = 8;
    int maxBlocks = 500000;
};
void DiffRun::run(void){
  if (nDiff < nTx){
    nTx = nDiff;
  }
  if (nAtoms < nTy) { 
    nTy = nAtoms;
  }
  int nBx = floor(nDiff / nTx);
  int nBy = floor(nAtoms / nTy);
  if (nBx > maxBlocks) {
    nBx = maxBlocks;
  }
  std::cout << "#Atoms: " << nAtoms << ", #Diffractions: " << nDiff << ", #AtomTypes: " << nAtomTypes << std::endl;
  std::cout << "Blocks: " << nBx << " x " << nBy << " Threads: " << nTx << " x " << nTy << std::endl;
  float memGuess = (nDiff*(3+4)*sizeof(double)+nAtoms*4*sizeof(double)+nAtoms*1*sizeof(int));
  std::cout << "MemUsage: " << memGuess*1e-6 << " MB" << std::endl;

  dim3 threadsPerBlock(nTx, nTy);
  dim3 numBlocks(nBx, nBy);

  gpuErrchk(hipMallocManaged(&atomicF, nAtomTypes*9*sizeof(double)));
  gpuErrchk(hipMallocManaged(&atomType, nAtoms*sizeof(int)));
  gpuErrchk(hipMallocManaged(&atomOccu, nAtoms*sizeof(double)));
  gpuErrchk(hipMallocManaged(&atomPos3D0, nAtoms*sizeof(double)));
  gpuErrchk(hipMallocManaged(&atomPos3D1, nAtoms*sizeof(double)));
  gpuErrchk(hipMallocManaged(&atomPos3D2, nAtoms*sizeof(double)));
  gpuErrchk(hipMallocManaged(&atomDW3D0, nAtoms*sizeof(double)));
  gpuErrchk(hipMallocManaged(&atomDW3D1, nAtoms*sizeof(double)));
  gpuErrchk(hipMallocManaged(&atomDW3D2, nAtoms*sizeof(double)));
  gpuErrchk(hipMallocManaged(&diffPos3D0, nDiff*sizeof(double)));
  gpuErrchk(hipMallocManaged(&diffPos3D1, nDiff*sizeof(double)));
  gpuErrchk(hipMallocManaged(&diffPos3D2, nDiff*sizeof(double)));
  gpuErrchk(hipMallocManaged(&diffFstaticRe, nDiff*sizeof(double)));
  gpuErrchk(hipMallocManaged(&diffFstaticIm, nDiff*sizeof(double)));
  gpuErrchk(hipMallocManaged(&diffFunitRe, nDiff*sizeof(double)));
  gpuErrchk(hipMallocManaged(&diffFunitIm, nDiff*sizeof(double)));

  calcF<<<numBlocks, threadsPerBlock>>>(
    nAtoms, nDiff, 
    diffPos3D0, diffPos3D1, diffPos3D2,
    atomOccu, atomType,
    atomPos3D0, atomPos3D1, atomPos3D2,
    atomDW3D0, atomDW3D1, atomDW3D2,
    atomicF,
    diffFunitRe, diffFunitIm);

  // Wait for GPU to finish before accessing on host
  gpuErrchk(hipDeviceSynchronize());
  finishedRun = true;
};


class DiffRunController 
{
  public:
    std::vector<DiffRun*> diffRunsAll;
    DiffRun *diffRunCurrent;

    void addRun(DiffRun *diffRunAdd);
    DiffRunController(DiffRun* diffRunInit);
    void runAll(void);
};
DiffRunController::DiffRunController(DiffRun* diffRunInit) {
  addRun(diffRunInit);
}
void DiffRunController::addRun(DiffRun* diffRunAdd) {
  diffRunAdd = diffRunAdd;
  diffRunsAll.push_back(diffRunAdd);
}
void DiffRunController::runAll(void) {
  for (std::vector<DiffRun*>::iterator diffRunThis = diffRunsAll.begin();
      diffRunThis != diffRunsAll.end(); ++diffRunThis) {
    diffRunCurrent = *diffRunThis;
    diffRunCurrent->run();

    // Test
    float output = 0.0;
    std::cout << std::fixed;
    std::cout.precision(2);
    for(int index = 0; index < 100; ++index)
    {
      output = (pow(diffRunCurrent->diffFunitRe[index], 2) + pow(diffRunCurrent->diffFunitIm[index], 2));
      std::cout << output << " ";
      if ((index+1) % 10 == 0) {
        std::cout << std::endl;
      }
    }
    std::cout << std::endl;
  }
}


int main(int argc, char** argv) 
{
  long nAtoms = atol(argv[1]); // along thread.y
  long nDiff = atol(argv[2]); // along thread.x
  int nAtomTypes = 3; // Fe, O, Ir

  // create GPU pointer
  double  *atomicF = new double[nAtomTypes * 9];
  int     *atomType = new int[nAtoms];
  double  *atomOccu = new double[nAtoms];
  double  *atomPos3D0 = new double[nAtoms];
  double  *atomPos3D1 = new double[nAtoms];
  double  *atomPos3D2 = new double[nAtoms];
  double  *atomDW3D0 = new double[nAtoms];
  double  *atomDW3D1 = new double[nAtoms];
  double  *atomDW3D2 = new double[nAtoms];
  double  *diffPos3D0 = new double[nDiff];
  double  *diffPos3D1 = new double[nDiff];
  double  *diffPos3D2 = new double[nDiff];
  double  *diffFstaticRe = new double[nDiff];
  double  *diffFstaticIm = new double[nDiff];
  double  *diffFunitRe = new double[nDiff];
  double  *diffFunitIm = new double[nDiff];

  // Generate some test data:
  std::cout << "Init data " << std::endl;
  int newAtomType[] = {0, 1, 1, 0, 0, 1, 1, 0};
  double newAtomOccu[] = {0.9, 1.0, 0.9, 1.0,
                          0.9, 1.0, 0.9, 1.0};
  double newAtomPos3D[8][3] = {{0.0,0.5,0.0}, {0.5,0.5,0.0},
                               {0.0,0.0,0.0}, {0.5,0.0,0.0},
                               {0.25,0.5,0.5},{0.75,0.5,0.5},
                               {0.25,0.0,0.5},{0.75,0.0,0.5}};
  double newAtomDW3D[8][3] = {{-1.0,0.0,0.0}, {-1.0,-20.0,-40.0},
                              {-10.0,-20.0,0.0}, {0.0,0.0,-0.0},
                              {0.0,0.0,0.0}, {0.0,0.0,0.0},
                              {0.0,0.0,0.0}, {0.0,0.0,0.0}};
  for (int index = 0; index < nAtoms; ++index)
  {
    atomType[index] = newAtomType[index];
    atomOccu[index] = newAtomOccu[index];
    atomPos3D0[index] = newAtomPos3D[index][0];
    atomPos3D1[index] = newAtomPos3D[index][1];
    atomPos3D2[index] = newAtomPos3D[index][2];
    atomDW3D0[index] = newAtomDW3D[index][0];
    atomDW3D1[index] = newAtomDW3D[index][1];
    atomDW3D2[index] = newAtomDW3D[index][2];
  };
  for(int index = 0; index < nDiff; ++index)
  {
    diffPos3D0[index] = floor((double)index / 10.);
    diffPos3D1[index] = ceil((double)index / 20.);
    diffPos3D2[index] = (0.1 * (index % 10));
    diffFstaticRe[index] = 0.0;
    diffFstaticIm[index] = 0.0;
    diffFunitRe[index] = 0.0;
    diffFunitIm[index] = 0.0;
  };

  // Run 
  std::cout << "Create Runs " << std::endl;
  
  DiffRun* diffRunFirst =  new DiffRun();
  diffRunFirst->nAtoms = nAtoms;
  diffRunFirst->nDiff = nDiff;
  diffRunFirst->nAtomTypes = nAtomTypes;
  diffRunFirst->atomicF = atomicF;
  diffRunFirst->atomType = atomType;
  diffRunFirst->atomOccu = atomOccu;
  diffRunFirst->atomPos3D0 = atomPos3D0;
  diffRunFirst->atomPos3D1 = atomPos3D1;
  diffRunFirst->atomPos3D2 = atomPos3D2;
  diffRunFirst->atomDW3D0 = atomDW3D0;
  diffRunFirst->atomDW3D1 = atomDW3D1;
  diffRunFirst->atomDW3D2 = atomDW3D2;
  diffRunFirst->diffPos3D0 = diffPos3D0;
  diffRunFirst->diffPos3D1 = diffPos3D1;
  diffRunFirst->diffPos3D2 = diffPos3D2;
  diffRunFirst->diffFstaticRe = diffFstaticRe;
  diffRunFirst->diffFstaticIm = diffFstaticIm;
  diffRunFirst->diffFunitRe = diffFunitRe;
  diffRunFirst->diffFunitIm = diffFunitIm;


  DiffRunController* diffRunController =  new DiffRunController(diffRunFirst);
  diffRunController->addRun(diffRunFirst);

  std::cout << "Start calcF " << std::endl;
  diffRunController->runAll();


  // // Test
  // DiffRun* diffRunCurrent = diffRunController->diffRunCurrent;
  // float output = 0.0;
  // std::cout << std::fixed;
  // std::cout.precision(2);
  // for(int index = 0; index < 100; ++index)
  // {
  //   output = (pow(diffRunCurrent->diffFunitRe[index], 2) + pow(diffRunCurrent->diffFunitIm[index], 2));
  //   std::cout << output << " ";
  //   if ((index+1) % 10 == 0) {
  //     std::cout << std::endl;
  //   }
  // }
  // std::cout << std::endl;
  
  std::cout << "Fin " << std::endl;
  return 0;
}
